

#include "Internal.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdlib.h>
#include <excpt.h>
//#include <winnt.h>


CUDA_CV_GLOBAL Global;
CUDA_CV_GLOBAL GetGlobal() { return Global; }



void cvInit(const int max_resotution_x, const int max_resolution_y, const int image_channels, const int max_gaussian_kernel_size)
{
    _Init(&Global, max_resotution_x, max_resolution_y, image_channels, max_gaussian_kernel_size);
}


void cvDispose()
{
	_Dispose(&Global);
}


void cvSetErrorCallback(ImageCudaErrorCallback call)
{
	SetErrorCallback(call);
}


//int filterException(int code, PEXCEPTION_POINTERS ex) {
//	std::cout << "Filtering " << std::hex << code << std::endl;
//	return EXCEPTION_EXECUTE_HANDLER;
//}

CUDA_CV_RESULT cvReduce3x(cvBYTE* data, int width, int height, int image_channels, int output_channels, cvBOOL use_single_buffer, cvBOOL result_to_local)
{
	/*__try
	{*/
		return _Reduce3x(data, width, height, image_channels, output_channels, use_single_buffer, result_to_local);
	//}
	//__except (EXCEPTION_EXECUTE_HANDLER)
	//{
	//	int code = GetExceptionCode();
	//	//PEXCEPTION_POINTERS re = GetExceptionInformation();
	//}
}

CUDA_CV_RESULT cvGaussianBlur(cvBYTE* image_data, int width, int height, int channels, int gauss_size, cvBOOL use_single_buffer, cvBOOL result_to_local)
{
	return convoluteGaussianBlur(image_data, width, height, channels, gauss_size, use_single_buffer, result_to_local);
}




